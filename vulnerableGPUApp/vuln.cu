/*
    Small Test program that shows a code reuse style attack using the Nvidia CUDA driver code that is loaded into every CUDA process.
    The buffer is filled with input from stdin, only stopping the read when \r\n\r\n is reached, thus allowing for buffer overflows.
    When using buf.bin, there is a jump to a gadget which overwriters (among others) R4, the register used for the return value of res.
    Afterwards normal execution is resumed but this now uses the wrong return value (0xdeadbeef).
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdint>

using namespace std;

__device__ __noinline__ void target_function(uint32_t* someIntArr, uint32_t index) {
    printf("Target at %p reached!\n", &target_function);
}

// function where we attempt to overwrite the return address
__device__ __noinline__ u_int32_t calling_function(uint32_t* input) {
    // Allocate a buffer which we will overflow to overwrite the return address
    u_int32_t buf[4000];

    if(input[0] == 0xdaedbeef) {
        printf("Recursion\n");
        return calling_function(&input[1]);
    }
    
    printf("target function at %p\n", &target_function);

    // Copy the buffer over from the input until we reach a \r\n\r\n
    int i = 0;
    while(input[i] != 0x0a0d0a0d) {
        printf("In loop: %d\n", i);
        uint32_t valOrig = buf[i];
        uint32_t valNew = input[i];
        printf("%d: %#08x => %#08x\n", i, valOrig, valNew);
        i++;
    }
    printf("Out of loop\n");

    u_int32_t res = 0;
    for(int i = 0; i < 400; i++) {
        printf("buf[%d]: %#08x\n", i, buf[i]);
        res += buf[i];
    }
    res += 69;
    return res; 
}

// Kernel that calls a function who's return address is written to the stack
__global__ __noinline__ void kernel_calling_function(uint32_t* input) {
    uint64_t someLocalBuffer[128]; 
    u_int32_t res = calling_function(input);
    input[0] = res;
    someLocalBuffer[0] = res;
    
    printf("res: %lu (%#08x), %p, %p\n", someLocalBuffer[0], (uint32_t) someLocalBuffer[0], input, &kernel_calling_function);
}

int main(int argc, char **argv) {
    uint32_t* buf = (u_int32_t *)malloc(8000 * sizeof(u_int32_t));
    for(int i = 0; i < 8000; i++) {
        buf[i] = 0;
    }

    char* buf_char = (char*)buf;

    // Read the buffer from stdin
    int i = 0;
    char c;
    do {
        scanf("%c", &c);
        buf_char[i] = c;
        i++;
    } while(i < 3 || buf[(i/4) - 1] != 0x0a0d0a0d);
    printf("Read %d bytes\n", i);

    uint32_t* d_buf;
    hipMalloc(&d_buf, 8000 * sizeof(u_int32_t));
    hipMemcpy(d_buf, buf, 8000 * sizeof(u_int32_t), hipMemcpyHostToDevice);

    printf("Main started\n");
	kernel_calling_function<<<1, 1>>>(d_buf);
    auto err = hipDeviceSynchronize();

    // Copy back the buffer
    hipMemcpy(buf, d_buf, 8000 * sizeof(u_int32_t), hipMemcpyDeviceToHost);
    printf("Result: %#08x\n", buf[0]);

    printf("Error: %d\n", err);
    printf("Error: %s\n", hipGetErrorName(err));
    printf("Error: %s\n", hipGetErrorString(err));
    printf("Main finished\n");
	return 0;
}